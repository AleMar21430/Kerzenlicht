
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addArrays(int* a, int* b, int* result, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		result[idx] = a[idx] + b[idx];
	}
}